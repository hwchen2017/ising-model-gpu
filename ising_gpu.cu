#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <unistd.h>
#include <random>
#include <complex>
#include <hiprand/hiprand.h>
#include <chrono>
#include <hip/hip_runtime.h>
// #include "cuda_marco.h"

using namespace std; 
double J1 = 1.0; 

const int THREAD = 128; 


__global__ void init_spins(signed char *lattice, const float* __restrict__ randval_d, const long long L )
{
    const long long tid = static_cast<long long> (blockDim.x) * blockIdx.x + threadIdx.x; 

    if(tid >= L*L/2) return ; 
 
    signed char val; 
    if(randval_d[tid] < 0.5f)
        val = -1;
    else 
        val = 1; 

    lattice[tid] = val; 

}


template<bool is_black>
__global__ void metropolis(signed char* lattice, const signed char* __restrict__ op_lattice, const float* __restrict__ randval_d,
                            const float beta, const long long nx, const long long ny)
{

    const long long tid = static_cast<long long>(blockDim.x)*blockIdx.x + threadIdx.x; 
    if(tid >= nx * ny) return ; 

    signed char spin = lattice[tid]; 

    const int i = tid / ny, j = tid % ny; 

    int down = ((i + 1)%nx) * ny + j; 
    int up = ((i-1+nx)%nx) * ny + j; 
    int left = i*ny + (j-1+ny)%ny; 
    int right = i*ny + (j+1)%ny; 
    int nearj; 

    if(is_black)
        nearj = i%2==1? right: left;
    else 
        nearj = i%2==1? left: right; 

    signed char nn_sum = op_lattice[down] + op_lattice[up] + op_lattice[tid] + op_lattice[nearj];
    float dE = 2.0f * nn_sum * spin;  
    float acceptance_ratio = exp(-beta * dE); 
    
    if(randval_d[tid] < acceptance_ratio)
        lattice[tid] = -spin; 

}



void update(signed char *lattice_black, signed char *lattice_white, float *randval_d, hiprandGenerator_t& gen, float temp, long long L)
{

	int blocks = (L*L + THREAD - 1)/THREAD; 

	hiprandGenerateUniform(gen, randval_d, L*L/2); 
	metropolis<true><<<blocks, THREAD>>>(lattice_black, lattice_white, randval_d, 1.0/temp, L, L/2); 

	hiprandGenerateUniform(gen, randval_d, L*L/2); 
    metropolis<false><<<blocks, THREAD>>>(lattice_white, lattice_black, randval_d, 1.0/temp, L, L/2);


}



void calculate_energy_mag(signed char* lattice_b_h, signed char* lattice_w_h, signed char* lattice_black, signed char* lattice_white, 
                        signed char* lattice_h, const long long L, double& energy, double& mag )
{
    
    hipMemcpy(lattice_b_h, lattice_black, L*L/2*sizeof(*lattice_b_h), hipMemcpyDeviceToHost); 
    hipMemcpy(lattice_w_h, lattice_white, L*L/2*sizeof(*lattice_w_h), hipMemcpyDeviceToHost);

    mag = 0.0; 
    energy = 0.0; 


    for(int i=0;i<L;i++)
        for(int  j=0;j<L/2;j++)
        {
            if(i%2)
            {
                lattice_h[i*L + 2*j+1] = lattice_b_h[i*L/2 + j]; 
                lattice_h[i*L + 2*j] = lattice_w_h[i*L/2 + j]; 
            }
            else 
            {
                lattice_h[i*L + 2*j] = lattice_b_h[i*L/2 + j]; 
                lattice_h[i*L + 2*j+1] = lattice_w_h[i*L/2 + j]; 
            }

            mag += (int)lattice_h[i*L+2*j]; 
            mag += (int)lattice_h[i*L+2*j+1]; 
        }

    mag /= (double)(L*L); 
    mag = fabs(mag); 

    int pos, up, down, left, right; 
    float nn_sum; 

    for(int i=0;i<L;i++)
        for(int j=0;j<L;j++)
        {
            pos = i*L + j; 
            up = ((i-1+L)%L)*L + j; 
            down = ((i+1)%L)*L + j; 
            left = i*L + (j-1+L)%L; 
            right = i*L + (j+1)&L; 

            nn_sum = lattice_h[up] +  lattice_h[down] + lattice_h[left] + lattice_h[right]; 

            energy += (float)lattice_h[pos] * nn_sum; 
        }

    energy *= -J1; 
    energy /= 2.0; 

    energy /= (double)(L*L); 

}


void save_spin_config(signed char* lattice_h, const long long L, string filename)
{

    ofstream fp; 
    fp.open(filename, ios::out); 

    for(int i=0;i<L;i++)
    {
        for(int j=0;j<L;j++)
            fp<<(int)lattice_h[i*L+j]<<" "; 
        fp<<endl; 
    }

    fp.close(); 
}



int main(int argc, char* argv[])
{

    std::random_device rd; 

    long long L = 1024;
    float temp = 2.0f;
    int MC_sweep = 200000;
    int MC_measure  = 100;  
    unsigned long long seed = (unsigned long long)rd(); 
    bool write_to_file = false; 


    char ch; 
    while((ch = getopt(argc, argv, "l:m:f:s:t:")) != EOF)
    {
        switch(ch)
        {
            case 'l' : L = atoi(optarg);
            break; 
            case 't' : temp = atof(optarg); 
            break; 
            case 's' : MC_sweep = atoi(optarg); 
            break;
            case 'm' : MC_measure = atoi(optarg); 
            break; 
            case 'f' : write_to_file = atoi(optarg); 
            break; 

        }
    }



    printf("\tlattice dimensions: %lld x %lld\n", L, L);
    printf("\tMC sweeps: %d\n", MC_sweep);


    hiprandGenerator_t gen; 
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10); 
    hiprandSetPseudoRandomGeneratorSeed(gen, seed); 

    float *randval_d;
    hipMalloc(&randval_d, L*L/2*sizeof(*randval_d)); 

    signed char *lattice_black, *lattice_white; 

    hipMalloc(&lattice_black, L*L/2*sizeof(*lattice_black)); 
    hipMalloc(&lattice_white, L*L/2*sizeof(*lattice_white)); 

    signed char* lattice_w_h, *lattice_b_h, *lattice_h; 

    lattice_h = (signed char*)malloc(L*L*sizeof(*lattice_h) ); 
    lattice_b_h = (signed char*)malloc(L*L/2*sizeof(*lattice_b_h)); 
    lattice_w_h = (signed char*)malloc(L*L/2*sizeof(*lattice_w_h)); 


    int blocks = (L*L/2 + THREAD -1)/THREAD; 

    hiprandGenerateUniform(gen, randval_d, L*L/2);
    init_spins<<<blocks, THREAD>>>(lattice_black, randval_d, L); 

    hiprandGenerateUniform(gen, randval_d, L*L/2); 
    init_spins<<<blocks, THREAD>>>(lattice_white, randval_d, L); 

    hipDeviceSynchronize(); 
    
    string filename, s_temp; 
    ofstream out; 

    if(write_to_file)
        out.open("energy_magnetization_" + to_string(L) + "x" + to_string(L) + "_" + to_string(MC_sweep) + ".txt"); 

    // for(int cnt = 300; cnt >= 150;)
    // {
        // temp = cnt * 0.01f; 
        
        // if(cnt > 250 or cnt <= 200) cnt -= 10; 
        // else cnt -= 5; 
        
        printf("Start equilibration for temperature %.2f\n", temp); 
          
        auto t0 = chrono::high_resolution_clock::now(); 

        //equilibrate the system to target temperature
        seed = (unsigned long long)rd(); 
        hiprandSetPseudoRandomGeneratorSeed(gen, seed); 


        for(int i=0;i<MC_sweep;i++)
        {
            update(lattice_black, lattice_white, randval_d, gen, temp, L); 

//             if(i%10000 == 0 or i == MC_sweep - 1)
//                 cout<<"Completed "<<i<<" MC sweep"<<endl; 
        }
        
        
        hipDeviceSynchronize();
        printf("Equilibration for temperature %.2f is done!\n", temp);  


        auto t1 = chrono::high_resolution_clock::now(); 
        auto elapsed  = t1 - t0; 
        double time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count(); 


        printf("Elapsed time for equilibrium: %.2f ms\n", time_ms);
        printf("MC sweep per ms: %f\n", (double) MC_sweep/time_ms );


        double energy = 0.0, mag = 0.0; 
        double avg_ene = 0.0, avg_mag = 0.0, ene_sq = 0.0, mag_sq = 0.0; 

        //collect meaurements for energy and magnetization
        for(int i=1;i<=MC_measure;i++)
        {
            for(int j=0;j<100;j++)
                update(lattice_black, lattice_white, randval_d, gen, temp, L); 

            hipDeviceSynchronize(); 
            calculate_energy_mag(lattice_b_h, lattice_w_h, lattice_black, lattice_white, lattice_h, L, energy, mag);
            avg_ene += energy; 
            avg_mag += mag; 
            ene_sq += norm(energy); 
            mag_sq += norm(mag);

        }

        avg_ene /= (double)MC_measure; 
        avg_mag /= (double)MC_measure; 
        ene_sq /= (double)MC_measure; 
        mag_sq /= (double)MC_measure; 
    
        printf("Energy: %.6f\n", avg_ene); 
        printf("Magnetization: %.2f\n\n", avg_mag);  
        
        if(write_to_file)
            out<<temp<<" "<<avg_ene<<" "<<ene_sq<<" "<<avg_mag<<" "<<mag_sq<<endl; 
    
        s_temp = to_string(temp); 
        while(s_temp.back() == '0') s_temp.pop_back(); 

        filename = "spin_config_" + to_string(L) + "x" + to_string(L) + "_" + to_string(MC_sweep) + "_temp_" + s_temp + ".txt";  


        if(write_to_file) save_spin_config(lattice_h, L, filename);
        
     
    // }
    

    out.close(); 

	return 0; 
}